#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"

#include <stdio.h>

#define BLOCK_SIZE (16)
#define RESULT_BLOCK_Y    (16)
#define SUBMATRIX_K   (4)
#define MAX_GRID_SIZE (65535) 
/******************************************************************************\
 * Section 1: Helper Structs                                                  *
\******************************************************************************/

/**
 * \brief Identity operation.
 * 
 * This struct contains the identity operation so this operation can be accessed
 * via template.
 */
struct Identity
{
    __device__
    static double func(double x) {return x;}
};

/**
 * \brief Sigmoid operation.
 * 
 * This struct contains the sigmoid operation so this operation can be accessed
 * via template.
 */
struct Sigmoid
{
    __device__
    static double func(double x){return 1 / (1 + exp(-x));}
};

/******************************************************************************\
 * Section 2: General Functions                                               *
\******************************************************************************/

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    return result;
}

__global__ void myGEMM_fast_kernel(double *A, double *B, double *C,
                                   double alpha, double beta,
                                   int M, int N, int K)
{
    // Step 1: Find the block root
    int i  =  blockDim.x*threadIdx.y + threadIdx.x;
    int i0 = (blockDim.x*blockDim.y)*blockIdx.x + i;
    int j0 = (RESULT_BLOCK_Y)*blockIdx.y;


    // Step 2: Initialize shared and register memory:
    __shared__ double B_block[SUBMATRIX_K*RESULT_BLOCK_Y];
    double A_block[SUBMATRIX_K];
    double C_out[RESULT_BLOCK_Y];

    // Step 3: Initialize C_out:
    for(int j = 0; j < RESULT_BLOCK_Y; j++)
        C_out[j] = 0;

    int num_iters = ((N - j0 < RESULT_BLOCK_Y) ? (N - j0) : RESULT_BLOCK_Y);
    int B_col = (i/SUBMATRIX_K) + j0;

    // Step 4: Iterate through all but the last blocks of A and B
    for(int k0 = 0; k0 < K; k0 += SUBMATRIX_K)
    {
        // Step 4a: Load the A block into shared memory
        if(i0 < M)
        {
            for(int k = 0; k < SUBMATRIX_K; k++)
            {
                if((k0 + k) < K)
                    A_block[k] = A[M*(k0 + k) + i0];
            }
        }

        __syncthreads();

        // Step 4b: Load the B block into shared memory
        if((B_col < N) && (k0 + (i % SUBMATRIX_K) < K))
        {
            B_block[i] = B[K*B_col + k0 + (i % SUBMATRIX_K)];
        }

        __syncthreads();

        // Step 4c: Compute the results:
        for(int j = 0; j < num_iters; j++)
        {
            for(int k = 0; k < SUBMATRIX_K; k++)
            {
                if((k0 + k) < K) 
                    C_out[j] += A_block[k] * B_block[SUBMATRIX_K*j + k];
            }
        }
    }
    // Step 5: Accumulate results in C
    for(int j = 0; j < num_iters; j++)
    {
        if(i0 < M)
        {
            C[(j0 + j)*M + i0] = alpha*C_out[j] + beta*C[(j0 + j)*M + i0]; 
        }
    }

}

template<bool IncludeOffset, bool TransposeA, bool TransposeB>
__global__ void
__launch_bounds__(256)
myGEMM_shared_kernel(double const * const __restrict__ A, 
                     double const * const __restrict__ B, 
                     double * __restrict__ C,
                     const double alpha, const double beta,
                     const int M, const int N, const int K)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    double old_val, product;

    __shared__ double A_submat[BLOCK_SIZE*(BLOCK_SIZE + 1)];
    __shared__ double B_submat[BLOCK_SIZE*(BLOCK_SIZE + 1)];


    product  = 0;
    old_val = (IncludeOffset ? C[row + M*col] : 0);
    
    for(int k0 = 0; k0 < K; k0 += blockDim.x)
    {
        __syncthreads();
        // Step 1: Update A_submat:
        if(TransposeA) 
        {
            A_submat[threadIdx.y + (blockDim.x + 1)*threadIdx.x] 
                = ((k0 + threadIdx.x < K) ?
                   ((row - threadIdx.x + threadIdx.y < M) ? 
                    A[(k0 + threadIdx.x) + 
                      K*(row - threadIdx.x + threadIdx.y)] : 0) : 0);
        } else {
            A_submat[threadIdx.x + (blockDim.x + 1)*threadIdx.y] 
                = ((k0 + threadIdx.y < K) ?
                   ((row < M) ? A[row + M*(k0 + threadIdx.y)] : 0) : 0);
        }
    
        // Step 2: Update B_submat:
        if(TransposeB)
        {
            B_submat[threadIdx.y + (blockDim.x+1)*threadIdx.x] 
                = ((k0 + threadIdx.y < K) ?
                   ((col - threadIdx.y + threadIdx.x < N) ? 
                    B[(col - threadIdx.y + threadIdx.x) + 
                      N*(k0 + threadIdx.y)] : 0) : 0);
            
        } else {
            B_submat[threadIdx.x + (blockDim.x+1)*threadIdx.y]
                = ((k0 + threadIdx.x < K) ? 
                ((col < N) ? B[(k0 + threadIdx.x) + K*col] : 0) : 0);
        }
        __syncthreads();
        
        // Step 3: Accumulate the results:
        int num_iters = min(K - k0, blockDim.x);
        for(int k = 0; k < num_iters; k++) 
        {
            product += A_submat[threadIdx.x + k*(blockDim.x+1)] *
                       B_submat[k + threadIdx.y*(blockDim.x+1)];
        }
    }
    if((row < M) && (col < N))
        C[row + M*col] = alpha*product + beta*old_val;
}


template<bool IncludeOffset, bool TransposeA, bool TransposeB>
__global__ void 
__launch_bounds__(256)
myGEMM_tile_kernel(double const * const __restrict__ A, 
                   double const * const __restrict__ B, 
                   double * __restrict__ C,
                   const double alpha, const double beta,
                   const int M, const int N, const int K)
{
    const int block_root_i0 = 64*blockIdx.x;
    const int block_root_j0 = 64*blockIdx.y;

    __shared__ double A_submat[65][4]; // TODO
    __shared__ double B_submat[5][64];

    double product[4][4] = {{0}};
    double A_frag[4];
    double B_frag[4];

    const int tile_root_i = (threadIdx.x/32) * 32 + 2*(threadIdx.x % 8);
    const int tile_root_j = threadIdx.y*16 + ((threadIdx.x % 32)/8) * 2;

    for(int k0 = 0; k0 < K; k0 += 4)
    {

        // Update the shared memory
        __syncthreads();
        if(TransposeA)
        {
            A_submat[threadIdx.x/4 + 16*threadIdx.y][threadIdx.x % 4] = 
              ((block_root_i0 + threadIdx.x/4 + 16*threadIdx.y < M) ? 
              ((k0 + (threadIdx.x % 4) < K) ?
               A[(k0 + (threadIdx.x % 4)) + K*(block_root_i0 + threadIdx.x/4 +
                   16*threadIdx.y)] : 0) : 0);

        } else {
            A_submat[threadIdx.x][threadIdx.y] = 
              ((block_root_i0 + threadIdx.x < M) ? ((k0 + threadIdx.y < K) ?
               A[(block_root_i0 + threadIdx.x) + M*(k0 + threadIdx.y)] : 0): 0);
        }

        if(TransposeB)
        {
            B_submat[threadIdx.y][threadIdx.x] = 
              ((block_root_j0 + threadIdx.x < N) ? ((k0 + threadIdx.y < K) ?
               B[(block_root_j0 + threadIdx.x) + N*(k0 + threadIdx.y)] : 0): 0);
        } else {
            B_submat[threadIdx.x % 4][threadIdx.x/4 + 16*threadIdx.y] =
              ((block_root_j0 + threadIdx.x/4 + 16*threadIdx.y < N) ? 
              ((k0 + (threadIdx.x % 4) < K) ?
               B[(k0 + (threadIdx.x % 4)) + K*(block_root_j0 + threadIdx.x/4 +
                   16*threadIdx.y)] : 0) : 0);
        }

        __syncthreads();


        // Loop through the tiles accumulating the product at each
#pragma unroll 3
        for(int k = 0; k < 4; k++)
        {
            // Step 1: Copy the fragments of A and B for this thread.
            for(int i = 0; i < 2; i++)
                for(int l = 0; l < 2; l++)
                {
                    A_frag[2*i + l] = A_submat[tile_root_i + 16*i + l][k];
                    B_frag[2*i + l] = B_submat[k][tile_root_j +  8*i + l];
                }
            // Step 2: Accumulate the product:
            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    product[i][j] += A_frag[i]*B_frag[j];
        }
    }
    // Now update C
#pragma unroll 4
    for(int i = 0; i < 2; i++)
      for(int j = 0; j < 2; j++)
        for(int l = 0; l < 2; l++)
          for(int m = 0; m < 2; m++)
          {
              if((block_root_i0 + tile_root_i + 16*i + l < M) &&
                 (block_root_j0 + tile_root_j + 8*j + m < N))
              {
                  C[(block_root_i0 + tile_root_i + 16*i + l) 
                    + M*(block_root_j0 + tile_root_j + 8*j + m)] =
                    alpha*product[2*i + l][2*j + m] +
                    (IncludeOffset ? 
                    beta *
                    C[(block_root_i0 + tile_root_i + 16*i + l) 
                      + M*(block_root_j0 + tile_root_j + 8*j + m)] : 0);
              }

          }
}


/**
 * \brief Kernel for in-place GEMM opration.
 *
 * See myGEMM for more details. We have used this Naive GEMM implementation for
 * the time being. The operation PostOp::func is applied to each element in C
 * after the product has been found. In some cases, the offset is not necessary
 * so we use set the IncludeOffset to false so we do not have to waste a memory
 * access to C. Furthermore, we have additional template arguments which tell us
 * whether to transpose matrix A before multiplication and whether to transpose
 * matrix B before multiplication.
 *
 * arguments:
 *     A, B, C, alpha, beta: Matrices and scalars in the general GEMM framework:
 *     C <- alpha*A*B + beta*C
 *
 *     M: Number of rows in A and C
 *     N: Number of columns in C and B
 *     K: Number of columns in A and rows in B
 */
template<class PostOp, bool IncludeOffset, bool TransposeA, bool TransposeB>
__global__ void myGEMM_kernel(double *A, double *B, double *C,
                              double alpha, double beta,
                              int M, int N, int K)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    // Number of rows and columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int num_cols = (N - col + (blockDim.y * gridDim.y - 1)) /
                   (blockDim.y * gridDim.y);

    int curr_row, curr_col;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;
        for(int j = 0; j < num_cols; j++)
        {
            curr_col = col + j*blockDim.y*gridDim.y;

            double product = 0;
            double old_val = IncludeOffset ? C[M*curr_col + curr_row] : 0;
            double Apart, Bpart;

            for(int k = 0; k < K; k++)
            {
               Apart = (TransposeA ? A[curr_row*K + k] : A[k*M + curr_row]);
               Bpart = (TransposeB ? B[k*N + curr_col] : B[curr_col*K + k]);
               product += Apart * Bpart;
            }

            C[M*curr_col + curr_row] = PostOp::func(
                                           alpha*product + beta*old_val);
        }
    }
}


/**
 * \brief Kernel for Matrix multiplication with vector accumulator.
 *
 * We have used this Naive GEMM implementation for the time being. The operation
 * PostOp::func is applied to each element in C after the product has been
 * found.
 *
 * Arguments:
 *     A, B, C, v, alpha, beta: Matrices, vectors, and scalars in the framework:
 *     C <- alpha*A*B + beta*[ v v v ... v]
 *
 *     M: Number of rows in A, C and v
 *     N: Number of columns in C and B and the matrix [ v v v ... v ]
 *     K: Number of columns in A and rows in B
 */
template<class PostOp>
__global__ void GEMM_vector_kernel(double *A, double *B, double *C, double *v,
                                   int M, int N, int K)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    // Number of rows and columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int num_cols = (N - col + (blockDim.y * gridDim.y - 1)) /
                   (blockDim.y * gridDim.y);

    int curr_row, curr_col;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;
        for(int j = 0; j < num_cols; j++)
        {
            curr_col = col + j*blockDim.y*gridDim.y;

            double product = 0;
            double old_val = v[curr_row];

            for(int k = 0; k < K; k++)
            {
               product += A[k*M + curr_row] * B[curr_col*K + k];
            }

            C[M*curr_col + curr_row] = 
                PostOp::func(product + old_val);
        }
    }
}

template<class PostOp>
__global__ void
__launch_bounds__(256)
GEMM_vector_shared_kernel(double *A, double *B, double *C, double *v,
                          int M, int N, int K)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    double old_val, product;

    __shared__ double A_submat[BLOCK_SIZE*(BLOCK_SIZE+1)];
    __shared__ double B_submat[BLOCK_SIZE*(BLOCK_SIZE+1)];


    product  = 0;
    old_val = v[row];
    
    for(int k0 = 0; k0 < K; k0 += blockDim.x)
    {
        __syncthreads();
        // Step 1: Update A_submat:
        A_submat[threadIdx.x + (blockDim.x+1)*threadIdx.y] 
            = ((k0 + threadIdx.y < K) ?
               ((row < M) ? A[row + M*(k0 + threadIdx.y)] : 0) : 0);
    
        // Step 2: Update B_submat:
        B_submat[threadIdx.x + (blockDim.x+1)*threadIdx.y]
            = ((k0 + threadIdx.x < K) ? 
            ((col < N) ? B[(k0 + threadIdx.x) + K*col] : 0) : 0);

        __syncthreads();
        
        // Step 3: Accumulate the results:
        int num_iters = min(K - k0, blockDim.x);
        for(int k = 0; k < num_iters; k++) 
        {
            product += A_submat[threadIdx.x + k*(blockDim.x+1)] *
                       B_submat[k + threadIdx.y*(blockDim.x+1)];
        }
    }
    if((row < M) && (col < N))
        C[row + M*col] = PostOp::func(product + old_val);
}


template<class PostOp>
__global__ void 
__launch_bounds__(256)
GEMM_vector_tile_kernel(double *A, double *B, double *C, double *v,
                                   int M, int N, int K)
{
    int block_root_i0 = 64*blockIdx.x;
    int block_root_j0 = 64*blockIdx.y;

    __shared__ double A_submat[65][4];
    __shared__ double B_submat[5][64];

    double product[4][4];
    double A_frag[4];
    double B_frag[4];

    for(int i = 0; i < 4; i++)
        for(int j = 0; j < 4; j++)
            product[i][j] = 0;

    int tile_root_i = (threadIdx.x/32) * 32 + 2*(threadIdx.x % 8);
    int tile_root_j = threadIdx.y*16 + ((threadIdx.x % 32)/8) * 2;

    for(int k0 = 0; k0 < K; k0 += 4)
    {

        // Update the shared memory
        __syncthreads();
        A_submat[threadIdx.x][threadIdx.y] = 
          ((block_root_i0 + threadIdx.x < M) ? ((k0 + threadIdx.y < K) ?
           A[(block_root_i0 + threadIdx.x) + M*(k0 + threadIdx.y)] : 0): 0);

        B_submat[threadIdx.x % 4][threadIdx.x/4 + 16*threadIdx.y] =
          ((block_root_j0 + threadIdx.x/4 + 16*threadIdx.y < N) ? 
          ((k0 + (threadIdx.x % 4) < K) ?
           B[(k0 + (threadIdx.x % 4)) + K*(block_root_j0 + threadIdx.x/4 +
               16*threadIdx.y)] : 0) : 0);
        __syncthreads();


        // Loop through the tiles accumulating the product at each
#pragma unroll 3
        for(int k = 0; k < 4; k++)
        {
            // Step 1: Copy the fragments of A and B for this thread.
            for(int i = 0; i < 2; i++)
                for(int l = 0; l < 2; l++)
                {
                    A_frag[2*i + l] = A_submat[tile_root_i + 16*i + l][k];
                    B_frag[2*i + l] = B_submat[k][tile_root_j +  8*i + l];
                }
            // Step 2: Accumulate the product:
            for(int i = 0; i < 4; i++)
                for(int j = 0; j < 4; j++)
                    product[i][j] += A_frag[i]*B_frag[j];
        }
    }
    // Now update C
#pragma unroll 4
    for(int i = 0; i < 2; i++)
      for(int j = 0; j < 2; j++)
        for(int l = 0; l < 2; l++)
          for(int m = 0; m < 2; m++)
          {
              if((block_root_i0 + tile_root_i + 16*i + l < M) &&
                 (block_root_j0 + tile_root_j + 8*j + m < N))
              {
                  C[(block_root_i0 + tile_root_i + 16*i + l) 
                    + M*(block_root_j0 + tile_root_j + 8*j + m)] =
                    PostOp::func(
                    product[2*i + l][2*j + m] +
                    v[(block_root_i0 + tile_root_i + 16*i + l)]); 
              }

          }
}

template<class PostOp, bool IncludeOffset, bool TransposeA, bool TransposeB,
         bool Vector>
void smart_GEMM_wrapper(double *A, double *B, double *C, double *v,
                        double alpha, double beta,
                        int M, int N, int K)
{
    dim3 blockSize (BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize  ((M + BLOCK_SIZE - 1)/BLOCK_SIZE, 
                    (N + BLOCK_SIZE - 1)/BLOCK_SIZE);
    if(Vector) 
    {
        if((M < 64) || (N < 64))
        {
            GEMM_vector_shared_kernel<PostOp><<<gridSize, blockSize>>>(
                    A, B, C, v, M, N, K);
        } else {
            blockSize.x = 64;
            blockSize.y = 4;
            gridSize.x  = (M + 63)/64;
            gridSize.y  = (N + 63)/64;
            GEMM_vector_tile_kernel<PostOp><<<gridSize, blockSize>>>(
                    A, B, C, v, M, N, K);
        }

    } else {
        if(false)
        {
            myGEMM_shared_kernel<IncludeOffset, TransposeA, 
                           TransposeB><<<gridSize, blockSize>>>(
                                   A, B, C, alpha, beta, M, N, K);
        } else {
            blockSize.x = 64;
            blockSize.y = 4;
            gridSize.x  = (M + 63)/64;
            gridSize.y  = (N + 63)/64;
            myGEMM_tile_kernel<IncludeOffset, TransposeA, 
                               TransposeB><<<gridSize, blockSize>>>(
                                   A, B, C, alpha, beta, M, N, K);
        }

    }
}


/*
 * \brief Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B +
 * beta*C. 
 * 
 * This function performs the in-place GEMM operation accelerated by the GPU.
 * The arguments are as follows:
 * 
 * A is an MxK matrix of doubles in col-major format.
 * B is an KxN matrix of doubles in col-major format.
 * C is an MxN matrix of doubles in col-major format.
 * alpha is the address of a scalar to multiply AB by before adding it to the
 * final result.
 * beta is the address of a scalar to multiply C by before adding alpha*AB to
 * it.
 *
 * Note that A, B, and C are pointers to device memory whereas alpha and beta
 * are pointers to host memory.
 */
int myGEMM(double* A, double* B, double* C, double* alpha, double* beta, int M,
           int N, int K) {

    smart_GEMM_wrapper<Identity, true, false, false, false>(
            A, B, C, NULL, *alpha, *beta, M, N, K);

    check_launch("myGEMM_kernel");

    return 0;
}




/******************************************************************************\
 * Section 3: Feed Forward Special Functions                                  *
\******************************************************************************/


/**
 * \brief Kernel for finding the softmax
 *
 * Although there's likely a smarter algorithm than this, we know that the
 * matrix we will be processing only has 10 rows for this use case, so we can
 * have a thread iterate through each row without too much of an issue.
 *
 * Furthermore, we store our results back in Z2 because we no longer need Z2
 * after we take the softmax, only yhat is neccesary in future steps.
 *
 * Z2 has L rows and N columns.
 *
 */
__global__ void softmax_kernel(double *Z2, int L, int N)
{
    int col = blockIdx.x*blockDim.x + threadIdx.x;


    // Number of columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_cols = (N - col + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);
    int curr_col;

    for(int c = 0; c < num_cols; c++)
    {
        curr_col = col + c*blockDim.x*gridDim.x;

        double sum = 0;
        for(int i = 0; i < L; i++)
            sum += exp(Z2[curr_col*L + i]);

        for(int i = 0; i < L; i++)
            Z2[curr_col*L + i] = exp(Z2[curr_col*L + i]) / sum;
    }

}

/**
 * \brief Wrapper to perform the feed-forward operation on device memory.
 *
 * arguments:
 *     d - A deviceCache (see gpu_func.h) containing pointers to all of the
 *         relevant neural network parameters.
 *     X - A pointer to a matrix of training images.
 *     N - The number of images (i.e. columns in X)
 */
int myFeedForward(deviceCache &d, double* X, int N)
{
    double *A1 = d.A1;
    double *W1 = d.W1;
    double *b1 = d.b1;

    double *A2 = d.A2;
    double *W2 = d.W2;
    double *b2 = d.b2;

    int K = d.K;
    int L = d.L;
    int M = d.M;

    // Step 1: First layer. We want to compute A1 which is M by N
    dim3 blockSize (64, 4);
    dim3 gridSize ((M + 63)/64, (N + 63)/64);

    // Use our vector-accumulating GEM to compute A1
    smart_GEMM_wrapper<Sigmoid, true, false, false, true>(
            W1, X, A1, b1, 1, 1, M, N, K);

    // Step 2a: Second layer. We want to compute A2. We start by storing Z2 in
    // the space of A2
    smart_GEMM_wrapper<Identity, true, false, false, true>(
            W2, A1, A2, b2, 1, 1, L, N, M);

    // Step 2b: Now we want to apply the softmax kernel to Z2 (which is stored
    // in A2) to get the correct A2 = yhat.
    blockSize.x = 256;
    blockSize.y = 1;
    gridSize.x  = std::min((int)((N + blockSize.x - 1)/blockSize.x),
                           MAX_GRID_SIZE);
    gridSize.y  = 1;
    softmax_kernel<<<gridSize, blockSize>>>(A2, L, N);

    return 0;
}

/**
 * \brief Computes the difference yhat - y in the back propogation and stores
 * the result in yhat.
 *  
 * Use blocks of 256x1 for this kernel.
 * 
 * arguments:
 *     yhat - The result of the neural network feed forward.
 *     y    - The correct results for each image
 *     L    - The number of rows in y and yhat
 *     N    - The number of columns in y and yhat
 */
__global__
void backPropDiff_kernel(double *yhat, double *y, int L, int N)
{
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Number of columns this thread must compute so that the whole matrix ends
    // up getting computed.
    int num_cols = (N - col + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int curr_col;

    for(int c = 0; c < num_cols; c++)
    {
        curr_col = col + c*blockDim.x*gridDim.x;
        for(int i = 0; i < L; i++)
            yhat[curr_col*L + i] = (yhat[curr_col*L + i] 
                                  - y[curr_col*L + i])/N;
    }
}


/**
 * \brief Scales a vector by an integer scale.
 *
 * This kernel is useful for scaling the gradients of b1 and b2 before they are
 * accumulated with gradients from other processes.
 * 
 * arguments:
 *     v     - pointer to the vector to scale
 *     M     - the number of rows in v
 *     scale - the integer to scale v by
 */
__global__
void vector_scale_kernel(double *v, int M, int scale)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;

    // Number of rows this thread must compute so that the whole matrix ends up
    // getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);
    int curr_row;
    
    for(int c = 0; c < num_rows; c++)
    {
        curr_row = row + c*blockDim.x*gridDim.x;
        v[curr_row] *= scale;
    }
}


/**
 * \brief Scales a matrix by an integer scale.
 *
 * This kernel is useful for scaling the gradients of W1 and W2 before they are
 * accumulated with gradients from other processes.
 * 
 * arguments:
 *     A     - pointer to the matrix to scale
 *     M     - the number of rows in A
 *     N     - the number of rows in A
 *     scale - the integer to scale v by
 */
__global__
void matrix_scale_kernel(double *A, int M, int N, int scale)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    // Number of rows and columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int num_cols = (N - col + (blockDim.y * gridDim.y - 1)) /
                   (blockDim.y * gridDim.y);

    int curr_row, curr_col;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;
        for(int j = 0; j < num_cols; j++)
        {
            curr_col = col + j*blockDim.y*gridDim.y;
            A[curr_row + M*curr_col] *= scale;
        }
    }
}

/**
 * \brief Kernel for summing by row to get a column vector in the first column
 * of the array A. Does only part of the sum: must be called multiply times (see
 * myRowSum below) to compute the row sum.
 *
 * arguments:
 *     A        - the array to sum the rows of
 *     M        - the number of rows in A
 *     N        - the number of columns in A
 *     stride   - the stride between adjacent terms summed by a single thread
 *     num_iter - the number of terms for each thread to sum
 */
__global__
void myRowSum_kernel(double *A, int M, int N, int stride, int num_iter)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;

    // The number of rows that this thread must compute to ensure all rows get
    // covered.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);
    int curr_row;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;

        // Compute the first and last column that this column includes in the
        // sum.
        int start_col = (blockIdx.y*blockDim.y + threadIdx.y)*stride*num_iter;
        int end_col   = (blockIdx.y*blockDim.y + threadIdx.y+1)*stride*num_iter;

        // Sum from the last column to the first with the appropriate stride.
        double sum = 0; 
        for(int col = end_col-stride; col >= start_col; col -= stride)
        {
            if(col < N)
                sum += A[col*M + curr_row];
        }
        if(start_col < N)
            A[start_col*M + curr_row] = sum;
    }
}


/**
 * \brief Kernel which performs the special Hadamard product present in the back
 * propogation.
 *
 * Note that dZ1 is stored in the memory occupied initially by dA1.
 *
 * arguments:
 *     dA1 - dA1 in the Hadamard product
 *     A1  - A1 in the Hadamard product
 *     M   - the number of rows in dA1 and A1
 *     N   - the number of columns in dA1 and A1
 */
__global__
void mySpecialHadamard_kernel(double *dA1, double *A1, 
                              int M, int N) 
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    // Number of rows and columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int num_cols = (N - col + (blockDim.y * gridDim.y - 1)) /
                   (blockDim.y * gridDim.y);

    int curr_row, curr_col;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;
        for(int j = 0; j < num_cols; j++)
        {
            curr_col = col + j*blockDim.y*gridDim.y;
            double A1_elem   =  A1[M*curr_col + curr_row];
            dA1[M*curr_col + curr_row] = dA1[M*curr_col + curr_row]
                                       * A1_elem * (1 - A1_elem);
        }
    }
}


/**
 * \brief Copies a matrix B to the area pointed to by A.
 *
 * arguments:
 *     A - the place to copy to
 *     B - the matrix to copy from
 *     M - the number of rows in A and B
 *     N - the number of columns in A and B
 */
__global__
void onDeviceCopy_kernel(double *A, double *B, int M, int N)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    // Number of rows and columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int num_cols = (N - col + (blockDim.y * gridDim.y - 1)) /
                   (blockDim.y * gridDim.y);

    int curr_row, curr_col;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;
        for(int j = 0; j < num_cols; j++)
        {
            curr_col = col + j*blockDim.y*gridDim.y;
            A[curr_col*M + curr_row] = B[curr_col*M + curr_row];
        }
    }
}

/**
 * \brief Wrapper which copies a matrix from B to A using onDeviceCopy_kernel.
 * 
 * arguments:
 *     A - the place to copy to
 *     B - the matrix to copy from
 *     M - the number of rows in A and B
 *     N - the number of columns in A and B
 */
void onDeviceCopy(double *A, double *B, int M, int N)
{
    dim3 blockSize (BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize (std::min((int)((M + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE),
                   std::min((int)((N + blockSize.y - 1)/blockSize.y), MAX_GRID_SIZE));

    onDeviceCopy_kernel<<<gridSize,blockSize>>>(A, B, M, N);
    //check_launch("onDeviceCopy kernel");
}



/**
 * \brief Sums by row to get a column vector in the first column of the array A.
 *
 * Uses myRowSum_kernel repeatedly to compute the row sum.
 *
 * arguments:
 *     A - matrix to sum the rows of
 *     M - number of rows in A
 *     N - number of columns in A
 */
void myRowSum(double *A, double *out, int M, int N)
{
    dim3 blockSize (BLOCK_SIZE, BLOCK_SIZE);

    // Define num_iters to ensure that our gridsize is not larger than the max
    // possible.
    int num_iters = std::max(4, (N + MAX_GRID_SIZE - 1) / MAX_GRID_SIZE);
    int stride = 1;
    
    dim3 gridSize (std::min((int)((M + blockSize.x - 1)/blockSize.x),
                   MAX_GRID_SIZE),
                   (N + (blockSize.y*stride*num_iters) - 1) /
                        (blockSize.y*stride*num_iters));


    // Call myRowSum_kernel repeatedly until all values have been summed into
    // the first row. We need to adjust the gridSize as we go because the number
    // of columns we need to sum is reduced on each iteration.
    myRowSum_kernel<<<gridSize, blockSize>>>(A, M, N, stride, num_iters);
    //check_launch("myRowSum_kernel");

    while (stride*num_iters < N)
    {
      stride *= num_iters;
      gridSize.y = (N + (blockSize.y*stride*num_iters) - 1) /
                        (blockSize.y*stride*num_iters);
      
      myRowSum_kernel<<<gridSize, blockSize>>>(A, M, N, stride, num_iters);
      //check_launch("myRowSum_kernel");
    }

    blockSize.x = 256;
    blockSize.y = 1;
    gridSize.x = std::min((int)((M + blockSize.x - 1)/blockSize.x),
                          MAX_GRID_SIZE);
    gridSize.y = 1;

    vector_scale_kernel<<<gridSize, blockSize>>>(A, M, N);
    //check_launch("vector_scale_kernel");

    onDeviceCopy_kernel<<<gridSize, blockSize>>>(out, A, M, 1);
    //check_launch("onDeviceCopy_kernel");
}









/**
 * \brief Function for carrying out the back propogation
 *
 * arguments:
 *     d   - a deviceCache (see gpu_func.h) containing pointers to all of the
 *           relevant neural network parameters
 *     X   - a pointer to a matrix of training images
 *     y   - a pointer to the matrix of image labels
 *     N   - the number of images (i.e. columns in X and y)
 *     reg - the regularization term
 */
int myBackPropogation(deviceCache &d, double *X, double *y, int N, double reg)
{
    // Set up aliases
    double *A1 = d.A1;
    double *W1 = d.W1;
    double *W2 = d.W2;
    double *dA1 = d.dA1;
    double *dW1 = d.dW1;
    double *dW2 = d.dW2;
    int L = d.L;
    int M = d.M;
    int K = d.K;

    double *diff = d.A2;
    double *dZ1 = d.dA1;

    // Step 1: Find the difference yhat - y
    dim3 blockSize (256, 1); 
    dim3 gridSize (std::min((int)((N + blockSize.x - 1)/blockSize.x),
                   MAX_GRID_SIZE), 1);

    backPropDiff_kernel<<<gridSize,blockSize>>>(diff, y, L, N);            
    //check_launch("backPropDiff kernel");

    // Step 2: Compute dW2
    onDeviceCopy(dW2, W2, L, M);

    // Include offset, transpose A1
    smart_GEMM_wrapper<Identity, true, false, true, false>(
            diff, A1, dW2, NULL, 1, reg, L, M, N);

    blockSize.x = BLOCK_SIZE;
    blockSize.y = BLOCK_SIZE;
    gridSize.x  = std::min((int)((L + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE);
    gridSize.y  = std::min((int)((M + blockSize.y - 1)/blockSize.y), MAX_GRID_SIZE);

    matrix_scale_kernel<<<gridSize, blockSize>>>(dW2, L, M, N);
    //check_launch("matrix_scale_kernel");

    // Step 3: Compute dA1
    // Do not include offset, transpose W2
    smart_GEMM_wrapper<Identity, false, true, false, false>(
        W2, diff, dA1, NULL, 1, 1, M, N, L);

    // Step 4: Compute db2
    myRowSum(diff, d.db2, L, N);

    // Step 5: Compute dZ1
    blockSize.x = BLOCK_SIZE;
    blockSize.y = BLOCK_SIZE;
    gridSize.x  = std::min((int)((M + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE);
    gridSize.y  = std::min((int)((N + blockSize.y - 1)/blockSize.y), MAX_GRID_SIZE);
    mySpecialHadamard_kernel<<<gridSize, blockSize>>>(dZ1, A1, M, N);      
    //check_launch("mySpecialHadamard_kernel");

    // Step 6: Compute dW1
    onDeviceCopy(dW1, W1, M, K);
    smart_GEMM_wrapper<Identity, true, false, true, false>(
            dZ1, X, dW1, NULL, 1, reg, M, K, N);


    blockSize.x = BLOCK_SIZE;
    blockSize.y = BLOCK_SIZE;
    gridSize.x  = std::min((int)((M + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE);
    gridSize.y  = std::min((int)((K + blockSize.y - 1)/blockSize.y), MAX_GRID_SIZE);

    matrix_scale_kernel<<<gridSize, blockSize>>>(dW1, M, K, N);

    // Step 7: Compute db1
    myRowSum(dZ1, d.db1, M, N);
    
    return 0;
}

/******************************************************************************\
 * Section 5: Gradient Descent Special Functions                              *
\******************************************************************************/

/**
 * \brief GPU kernel for performing the gradient descent update.
 *
 * The normalization factor allows us to renormalize the gradients after all of
 * our processes exchange and sum gradients. 
 *
 * arguments:
 *     W_or_b        - the parameter that we subtracting the gradient from
 *     grad          - the gradient that we are subtracting from W_or_b
 *     learning_rate - the learning rate
 *     M             - the number of rows in W_or_b
 *     N             - the number of columns in W_or_b
 *     normalization - a normalization integer to divide the gradeint by.
 */
__global__
void grad_descent_kernel(double *W_or_b, double *grad, double learning_rate,
                         int M, int N, int normalization)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    // Number of rows and columns this thread must compute so that the whole
    // matrix ends up getting computed.
    int num_rows = (M - row + (blockDim.x * gridDim.x - 1)) /
                   (blockDim.x * gridDim.x);

    int num_cols = (N - col + (blockDim.y * gridDim.y - 1)) /
                   (blockDim.y * gridDim.y);

    int curr_row, curr_col;

    for(int i = 0; i < num_rows; i++)
    {
        curr_row = row + i*blockDim.x*gridDim.x;
        for(int j = 0; j < num_cols; j++)
        {
            curr_col = col + j*blockDim.y*gridDim.y;
            W_or_b[curr_col*M + curr_row] -= learning_rate 
                * (grad[curr_col*M + curr_row]/normalization);
        }
    }
}

/**
 * \brief Performs the gradient descent update accelrated by the gpu.
 * 
 * arguments:
 *     d             - a deviceCache (see gpu_func.h) containing pointers to 
 *                     all of the
 *     N             - the number of images in the whole batch
 *     learning_rate - the learning rate for this training
 */
void myGradientDescent(deviceCache &d, double learning_rate, int N)
{
    // Step 1: Update W1
    dim3 blockSize (BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize  (std::min((int)((d.M + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE),
                    std::min((int)((d.K + blockSize.y - 1)/blockSize.y), MAX_GRID_SIZE)); 
    grad_descent_kernel<<<gridSize, blockSize>>>(d.W1, d.dW1, learning_rate,
                                                 d.M, d.K, N);
    //check_launch("grad_descent_kernel");

    // Step 2: Update W2
    gridSize.x = std::min((int)((d.L + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE);
    gridSize.y = std::min((int)((d.M + blockSize.y - 1)/blockSize.y), MAX_GRID_SIZE);
    grad_descent_kernel<<<gridSize, blockSize>>>(d.W2, d.dW2, learning_rate,
                                                 d.L, d.M, N);
    //check_launch("grad_descent_kernel");

    // Step 3: Update b1
    blockSize.x = 256;
    blockSize.y = 1;
    gridSize.x = std::min((int)((d.M + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE);
    gridSize.y = 1;
    grad_descent_kernel<<<gridSize, blockSize>>>(d.b1, d.db1, learning_rate,
                                                 d.M, 1, N);
    //check_launch("grad_descent_kernel");


    // Step 4: Update b2
    gridSize.x = std::min((int)((d.L + blockSize.x - 1)/blockSize.x), MAX_GRID_SIZE);
    grad_descent_kernel<<<gridSize, blockSize>>>(d.b2, d.db2, learning_rate,
                                                 d.L, 1, N);

    //check_launch("grad_descent_kernel");
}
